#include "hip/hip_runtime.h"
#include "compressor.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <nvcomp/zstd.hpp>

// CUDA 에러 체크 헬퍼 함수
#define CUDA_CHECK(err) { \
    hipError_t e = (err); \
    if (e != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(e) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; \
        throw std::runtime_error(hipGetErrorString(e)); \
    } \
}

bool compress_safetensor(
    const std::string& json_header,
    const std::vector<char>& tensor_data,
    CompressionResult& result,
    int compression_level)
{

    try {
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreate(&stream));

        { // 매니저 수명 관리를 위한 새 스코프
            const size_t internal_uncomp_chunk = 64 * 1024; // 64KB 권장
            nvcomp::ZstdManager manager(
                internal_uncomp_chunk,
                nvcompBatchedZstdCompressDefaultOpts,
                nvcompBatchedZstdDecompressDefaultOpts,
                stream);

            // 1) JSON 헤더 압축 (빈 헤더는 건너뜀)
            result.compressed_header.clear();
            if (!json_header.empty()) {
                void* d_uncompressed_header = nullptr;
                CUDA_CHECK(hipMalloc(&d_uncompressed_header, json_header.size()));
                CUDA_CHECK(hipMemcpyAsync(d_uncompressed_header, json_header.data(), json_header.size(), hipMemcpyHostToDevice, stream));

                auto header_comp_config = manager.configure_compression(json_header.size());

                void* d_compressed_header = nullptr;
                CUDA_CHECK(hipMalloc(&d_compressed_header, header_comp_config.max_compressed_buffer_size));

                manager.compress(
                    reinterpret_cast<const uint8_t*>(d_uncompressed_header),
                    reinterpret_cast<uint8_t*>(d_compressed_header),
                    header_comp_config);

                // 압축 완료 보장 후 크기 조회
                CUDA_CHECK(hipStreamSynchronize(stream));

                size_t actual_header_comp_size =
                    manager.get_compressed_output_size(reinterpret_cast<const uint8_t*>(d_compressed_header));
                result.compressed_header.resize(actual_header_comp_size);

                // 동기 복사(추가 동기화 불필요)
                CUDA_CHECK(hipMemcpy(result.compressed_header.data(),
                                      d_compressed_header,
                                      actual_header_comp_size,
                                      hipMemcpyDeviceToHost));

                CUDA_CHECK(hipFree(d_uncompressed_header));
                CUDA_CHECK(hipFree(d_compressed_header));

                std::cout << "JSON header compressed (GPU): " << json_header.size()
                          << " -> " << actual_header_comp_size << " bytes" << std::endl;
            }

            // 2) 텐서 데이터 GPU 압축 (빈 입력은 건너뜀)
            result.compressed_tensors.clear();
            result.chunk_info.clear();

            if (!tensor_data.empty()) {
                const size_t chunk_size = 1024ULL * 1024ULL * 64ULL; // 64MB
                size_t num_chunks = (tensor_data.size() + chunk_size - 1) / chunk_size;
                std::cout << "Starting tensor compression with " << num_chunks
                          << " chunks on GPU using nvCOMP 5.0..." << std::endl;

                const size_t max_input_chunk = std::min(chunk_size, tensor_data.size());

                // 디바이스 버퍼를 반복 사용(과대할당 방지)
                void* d_uncompressed_chunk = nullptr;
                CUDA_CHECK(hipMalloc(&d_uncompressed_chunk, max_input_chunk));

                auto comp_config_template = manager.configure_compression(max_input_chunk);
                void* d_compressed_chunk = nullptr;
                CUDA_CHECK(hipMalloc(&d_compressed_chunk, comp_config_template.max_compressed_buffer_size));

                // 호스트 임시 버퍼(페이지드). 필요 시 hipHostAlloc으로 변경 가능
                std::vector<char> host_comp_buf(comp_config_template.max_compressed_buffer_size);

                const char* current_tensor_ptr = tensor_data.data();
                size_t total_compressed_size = 0;

                for (size_t i = 0; i < num_chunks; ++i) {
                    const size_t current_chunk_size =
                        (i == num_chunks - 1)
                            ? (tensor_data.size() - i * chunk_size)
                            : chunk_size;

                    CUDA_CHECK(hipMemcpyAsync(d_uncompressed_chunk,
                                               current_tensor_ptr,
                                               current_chunk_size,
                                               hipMemcpyHostToDevice,
                                               stream));

                    // 청크별 압축 설정
                    auto comp_config = manager.configure_compression(current_chunk_size);

                    // 압축 실행
                    manager.compress(
                        reinterpret_cast<const uint8_t*>(d_uncompressed_chunk),
                        reinterpret_cast<uint8_t*>(d_compressed_chunk),
                        comp_config);

                    // 압축 완료 후 실제 크기 조회
                    CUDA_CHECK(hipStreamSynchronize(stream));
                    const size_t actual_comp_size =
                        manager.get_compressed_output_size(reinterpret_cast<const uint8_t*>(d_compressed_chunk));

                    // 동기 복사로 호스트에 수신
                    CUDA_CHECK(hipMemcpy(host_comp_buf.data(),
                                          d_compressed_chunk,
                                          actual_comp_size,
                                          hipMemcpyDeviceToHost));

                    // 결과 누적
                    result.compressed_tensors.insert(result.compressed_tensors.end(),
                                                     host_comp_buf.begin(),
                                                     host_comp_buf.begin() + static_cast<std::ptrdiff_t>(actual_comp_size));
                    result.chunk_info.push_back({ current_chunk_size, actual_comp_size });
                    total_compressed_size += actual_comp_size;

                    current_tensor_ptr += current_chunk_size;
                }

                CUDA_CHECK(hipFree(d_uncompressed_chunk));
                CUDA_CHECK(hipFree(d_compressed_chunk));

                std::cout << "Tensor data compressed (GPU): " << tensor_data.size()
                          << " -> " << total_compressed_size << " bytes" << std::endl;
            }
        } // 스트림 파괴 전에 매니저가 먼저 소멸됨

        CUDA_CHECK(hipStreamDestroy(stream));
    }
    catch (const std::exception& e) {
        std::cerr << "An error occurred during GPU compression: " << e.what() << std::endl;
        return false;
    }
    return true;
}

bool decompress_kang(
    const std::vector<char>& compressed_header,
    const std::vector<char>& compressed_tensors,
    const std::vector<std::pair<size_t, size_t>>& chunk_info,
    std::string& json_header,
    std::vector<char>& tensor_data)
{
    try {
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreate(&stream));

        { // 매니저 관리를 위한 새 스코프
            const size_t internal_uncomp_chunk = 64 * 1024;
            nvcomp::ZstdManager manager(
                internal_uncomp_chunk,
                nvcompBatchedZstdCompressDefaultOpts,
                nvcompBatchedZstdDecompressDefaultOpts,
                stream);

            // 1) JSON 헤더 해제 (빈 헤더는 건너뜀)
            json_header.clear();
            if (!compressed_header.empty()) {
                void* d_compressed_header = nullptr;
                CUDA_CHECK(hipMalloc(&d_compressed_header, compressed_header.size()));
                CUDA_CHECK(hipMemcpyAsync(d_compressed_header, compressed_header.data(), compressed_header.size(), hipMemcpyHostToDevice, stream));

                auto header_decomp_config =
                    manager.configure_decompression(reinterpret_cast<const uint8_t*>(d_compressed_header));

                void* d_decompressed_header = nullptr;
                CUDA_CHECK(hipMalloc(&d_decompressed_header, header_decomp_config.decomp_data_size));

                manager.decompress(
                    reinterpret_cast<uint8_t*>(d_decompressed_header),
                    reinterpret_cast<const uint8_t*>(d_compressed_header),
                    header_decomp_config);

                // 해제 완료 보장
                CUDA_CHECK(hipStreamSynchronize(stream));

                json_header.resize(header_decomp_config.decomp_data_size);
                // 동기 복사(추가 동기화 불필요)
                CUDA_CHECK(hipMemcpy(&json_header[0],
                                      d_decompressed_header,
                                      header_decomp_config.decomp_data_size,
                                      hipMemcpyDeviceToHost));

                CUDA_CHECK(hipFree(d_compressed_header));
                CUDA_CHECK(hipFree(d_decompressed_header));
            }

            // 2) 텐서 데이터 해제 (청크가 없으면 건너뜀)
            if (chunk_info.empty()) {
                tensor_data.clear();
            } else {
                size_t total_decompressed_size = 0;
                size_t max_original_size = 0;
                size_t max_compressed_size = 0;
                for (const auto& info : chunk_info) {
                    total_decompressed_size += info.first;
                    if (info.first > max_original_size) max_original_size = info.first;
                    if (info.second > max_compressed_size) max_compressed_size = info.second;
                }
                tensor_data.resize(total_decompressed_size);

                std::cout << "Starting tensor decompression for " << chunk_info.size()
                          << " chunks on GPU using nvCOMP 5.0..." << std::endl;

                const char* current_compressed_ptr = compressed_tensors.data();
                char* current_decompressed_ptr = tensor_data.data();

                // 디바이스 버퍼 재사용(0 크기 방지)
                void* d_compressed_chunk = nullptr;
                CUDA_CHECK(hipMalloc(&d_compressed_chunk, max_compressed_size));
                void* d_decompressed_chunk = nullptr;
                CUDA_CHECK(hipMalloc(&d_decompressed_chunk, max_original_size));

                for (const auto& info : chunk_info) {
                    const size_t original_size = info.first;
                    const size_t compressed_size = info.second;

                    CUDA_CHECK(hipMemcpyAsync(d_compressed_chunk,
                                               current_compressed_ptr,
                                               compressed_size,
                                               hipMemcpyHostToDevice,
                                               stream));

                    // 해제 설정(디바이스에서 헤더 읽음)
                    auto decomp_config =
                        manager.configure_decompression(reinterpret_cast<const uint8_t*>(d_compressed_chunk));

                    // 검증: 예상 해제 크기 확인
                    if (decomp_config.decomp_data_size != original_size) {
                        CUDA_CHECK(hipStreamSynchronize(stream));
                        CUDA_CHECK(hipFree(d_compressed_chunk));
                        CUDA_CHECK(hipFree(d_decompressed_chunk));
                        throw std::runtime_error("Decompressed size mismatch for chunk.");
                    }

                    manager.decompress(
                        reinterpret_cast<uint8_t*>(d_decompressed_chunk),
                        reinterpret_cast<const uint8_t*>(d_compressed_chunk),
                        decomp_config);

                    // 해제 완료 보장
                    CUDA_CHECK(hipStreamSynchronize(stream));

                    // 동기 복사로 호스트에 수신
                    CUDA_CHECK(hipMemcpy(current_decompressed_ptr,
                                          d_decompressed_chunk,
                                          original_size,
                                          hipMemcpyDeviceToHost));

                    current_compressed_ptr += compressed_size;
                    current_decompressed_ptr += original_size;
                }

                CUDA_CHECK(hipFree(d_compressed_chunk));
                CUDA_CHECK(hipFree(d_decompressed_chunk));
            }
        } // 스트림 파괴 전에 매니저가 먼저 소멸

        CUDA_CHECK(hipStreamDestroy(stream));
    }
    catch (const std::exception& e) {
        std::cerr << "An error occurred during GPU decompression: " << e.what() << std::endl;
        return false;
    }
    return true;
}


